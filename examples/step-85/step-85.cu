#include "hip/hip_runtime.h"
/* ---------------------------------------------------------------------
 *
 * Copyright (C) 2018 by the deal.II authors
 *
 * This file is part of the deal.II library.
 *
 * The deal.II library is free software; you can use it, redistribute
 * it, and/or modify it under the terms of the GNU Lesser General
 * Public License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 * The full text of the license can be found in the file LICENSE.md at
 * the top level directory of deal.II.
 *
 * ---------------------------------------------------------------------

 *
 * Authors: Bruno Turcksin, Oak Ridge National Laboratory, 2019
 */

// First include the necessary files from the deal.II libary.
#include <deal.II/base/conditional_ostream.h>
#include <deal.II/base/quadrature_lib.h>

#include <deal.II/dofs/dof_tools.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/solver_cg.h>
#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/precondition.h>

#include <deal.II/fe/fe_q.h>

#include <deal.II/grid/tria.h>
#include <deal.II/grid/grid_generator.h>

#include <deal.II/numerics/data_out.h>
#include <deal.II/numerics/vector_tools.h>

// This includes the data structures for the implementation of matrix-free
// methods on GPU
#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/matrix_free/cuda_matrix_free.h>
#include <deal.II/matrix_free/cuda_fe_evaluation.h>
#include <deal.II/matrix_free/cuda_matrix_free.h>

#include <fstream>

namespace Step85
{
  using namespace dealii;

  template <int dim, int fe_degree>
  class HelmholtzOperatorQuad
  {
  public:
    __device__ void
    operator()(CUDAWrappers::FEEvaluation<dim, fe_degree> *fe_eval,
               const unsigned int                          q) const;
  };



  template <int dim, int fe_degree>
  __device__ void HelmholtzOperatorQuad<dim, fe_degree>::
                  operator()(CUDAWrappers::FEEvaluation<dim, fe_degree> *fe_eval,
             const unsigned int                          q) const
  {
    fe_eval->submit_value(10. * fe_eval->get_value(q), q);
    fe_eval->submit_gradient(fe_eval->get_gradient(q), q);
  }



  template <int dim, int fe_degree>
  class LocalHelmholtzOperator
  {
  public:
    __device__ void operator()(
      const unsigned int                                          cell,
      const typename CUDAWrappers::MatrixFree<dim, double>::Data *gpu_data,
      CUDAWrappers::SharedData<dim, double> *                     shared_data,
      const double *                                              src,
      double *                                                    dst) const;

    static const unsigned int n_dofs_1d = fe_degree + 1;
    static const unsigned int n_local_dofs =
      dealii::Utilities::pow(fe_degree + 1, dim);
    static const unsigned int n_q_points =
      dealii::Utilities::pow(fe_degree + 1, dim);
  };



  template <int dim, int fe_degree>
  __device__ void LocalHelmholtzOperator<dim, fe_degree>::operator()(
    const unsigned int                                          cell,
    const typename CUDAWrappers::MatrixFree<dim, double>::Data *gpu_data,
    CUDAWrappers::SharedData<dim, double> *                     shared_data,
    const double *                                              src,
    double *                                                    dst) const
  {
    CUDAWrappers::FEEvaluation<dim, fe_degree, fe_degree + 1, 1, double>
      fe_eval(cell, gpu_data, shared_data);
    fe_eval.read_dof_values(src);
    fe_eval.evaluate(true, true);
    fe_eval.apply_quad_point_operations(
      HelmholtzOperatorQuad<dim, fe_degree>());
    fe_eval.integrate(true, true);
    fe_eval.distribute_local_to_global(dst);
  }



  template <int dim, int fe_degree>
  class HelmholtzOperator
  {
  public:
    HelmholtzOperator(const DoFHandler<dim> &          dof_handler,
                      const AffineConstraints<double> &constraints);

    // TODO add varying coefficient using a lambda function
    // void evaluate_coefficient(const Coefficient<dim> &coefficient_function);

    //    void compute_inverse_diagonal();

    void
    vmult(LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> &dst,
          const LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA>
            &src) const;

  private:
    CUDAWrappers::MatrixFree<dim, double> mf_data;
  };



  template <int dim, int fe_degree>
  HelmholtzOperator<dim, fe_degree>::HelmholtzOperator(
    const DoFHandler<dim> &          dof_handler,
    const AffineConstraints<double> &constraints)
  {
    MappingQGeneric<dim> mapping(fe_degree);
    typename CUDAWrappers::MatrixFree<dim, double>::AdditionalData
      additional_data;
    additional_data.mapping_update_flags = update_values | update_gradients |
                                           update_JxW_values |
                                           update_quadrature_points;
    const QGauss<1> quad(fe_degree + 1);
    mf_data.reinit(mapping, dof_handler, constraints, quad, additional_data);
  }



  template <int dim, int fe_degree>
  void HelmholtzOperator<dim, fe_degree>::vmult(
    LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> &      dst,
    const LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> &src)
    const
  {
    dst = 0.;
    LocalHelmholtzOperator<dim, fe_degree> helmholtz_operator;
    mf_data.cell_loop(helmholtz_operator, src, dst);
    mf_data.copy_constrained_values(src, dst);
  }



  template <int dim, int fe_degree>
  class HelmholtzProblem
  {
  public:
    HelmholtzProblem();
    ~HelmholtzProblem();

    void run();

  private:
    void setup_system();
    // TODO just do it on the host and then move to the GPU
    void assemble_rhs();
    void solve();
    void refine_grid();
    void output_results(const unsigned int cycle) const;

    MPI_Comm mpi_communicator;

    parallel::distributed::Triangulation<dim> triangulation;

    DoFHandler<dim> dof_handler;
    FE_Q<dim>       fe;

    IndexSet locally_owned_dofs;
    IndexSet locally_relevant_dofs;

    AffineConstraints<double>                          constraints;
    std::unique_ptr<HelmholtzOperator<dim, fe_degree>> system_matrix_dev;

    LinearAlgebra::distributed::Vector<double, MemorySpace::Host> solution_host;
    LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> solution_dev;
    LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA>
      system_rhs_dev;

    ConditionalOStream pcout;
  };



  template <int dim, int fe_degree>
  HelmholtzProblem<dim, fe_degree>::HelmholtzProblem()
    : mpi_communicator(MPI_COMM_WORLD)
    , triangulation(mpi_communicator)
    , dof_handler(triangulation)
    , fe(fe_degree)
    , pcout(std::cout, Utilities::MPI::this_mpi_process(mpi_communicator) == 0)
  {}



  template <int dim, int fe_degree>
  HelmholtzProblem<dim, fe_degree>::~HelmholtzProblem()
  {
    dof_handler.clear();
  }



  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::setup_system()
  {
    dof_handler.distribute_dofs(fe);

    locally_owned_dofs = dof_handler.locally_owned_dofs();
    DoFTools::extract_locally_relevant_dofs(dof_handler, locally_relevant_dofs);
    system_rhs_dev.reinit(locally_owned_dofs, mpi_communicator);

    constraints.clear();
    constraints.reinit(locally_relevant_dofs);
    DoFTools::make_hanging_node_constraints(dof_handler, constraints);
    VectorTools::interpolate_boundary_values(dof_handler,
                                             0,
                                             Functions::ZeroFunction<dim>(),
                                             constraints);
    constraints.close();
    system_matrix_dev.reset(
      new HelmholtzOperator<dim, fe_degree>(dof_handler, constraints));

    solution_host.reinit(locally_owned_dofs, mpi_communicator);
    solution_dev.reinit(locally_owned_dofs, mpi_communicator);
    system_rhs_dev.reinit(locally_owned_dofs, mpi_communicator);
  }



  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::assemble_rhs()
  {
    system_rhs_dev.add(1.);
  }



  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::solve()
  {
    PreconditionIdentity preconditioner;

    SolverControl solver_control(100, 1e-12 * system_rhs_dev.l2_norm());
    SolverCG<LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA>> cg(
      solver_control);
    cg.solve(*system_matrix_dev, solution_dev, system_rhs_dev, preconditioner);

    // Copy the solution from the device to the host
    LinearAlgebra::ReadWriteVector<double> rw_vector(locally_owned_dofs);
    rw_vector.import(solution_dev, VectorOperation::insert);
    solution_host.import(rw_vector, VectorOperation::insert);

    constraints.distribute(solution_host);
  }



  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::output_results(
    const unsigned int cycle) const
  {
    DataOut<dim> data_out;

    LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
      ghost_solution_host(locally_owned_dofs,
                          locally_relevant_dofs,
                          mpi_communicator);
    ghost_solution_host = solution_host;
    data_out.attach_dof_handler(dof_handler);
    data_out.add_data_vector(ghost_solution_host, "solution");
    data_out.build_patches();

    std::ofstream output(
      "solution-" + std::to_string(cycle) + "." +
      std::to_string(Utilities::MPI::this_mpi_process(mpi_communicator)) +
      ".vtu");
    DataOutBase::VtkFlags flags;
    flags.compression_level = DataOutBase::VtkFlags::best_speed;
    data_out.set_flags(flags);
    data_out.write_vtu(output);

    if (Utilities::MPI::this_mpi_process(mpi_communicator) == 0)
      {
        std::vector<std::string> filenames;
        for (unsigned int i = 0;
             i < Utilities::MPI::n_mpi_processes(mpi_communicator);
             ++i)
          filenames.emplace_back("solution-" + std::to_string(cycle) + "." +
                                 std::to_string(i) + ".vtu");

        std::string master_name =
          "solution-" + Utilities::to_string(cycle) + ".pvtu";
        std::ofstream master_output(master_name);
        data_out.write_pvtu_record(master_output, filenames);
      }
  }



  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::run()
  {
    for (unsigned int cycle = 0; cycle < 9 - dim; ++cycle)
      {
        pcout << "Cycle " << cycle << std::endl;

        if (cycle == 0)
          {
            GridGenerator::hyper_cube(triangulation, 0., 1.);
            triangulation.refine_global(3 - dim);
          }
        triangulation.refine_global(1);
        setup_system();
        assemble_rhs();
        solve();
        output_results(cycle);
        pcout << std::endl;
      }
  }
} // namespace Step85

int main(int argc, char *argv[])
{
  try
    {
      using namespace Step85;

      Utilities::MPI::MPI_InitFinalize mpi_init(argc, argv, 1);

      // By default, all the ranks will try to access the device 0.
      // If we are running with MPI support it is better to address different
      // graphic cards for different processes even if only one node is used.
      // The choice below is based on the MPI proccess id. MPI needs to be
      // initialized before using this function.
      int         n_devices       = 0;
      hipError_t cuda_error_code = hipGetDeviceCount(&n_devices);
      AssertCuda(cuda_error_code);
      const unsigned int my_id =
        Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
      const int device_id = my_id % n_devices;
      cuda_error_code     = hipSetDevice(device_id);
      AssertCuda(cuda_error_code);

      HelmholtzProblem<3, 3> helmhotz_problem;
      helmhotz_problem.run();
    }
  catch (std::exception &exc)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Exception on processing: " << std::endl
                << exc.what() << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  catch (...)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Unknown exception!" << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }

  return 0;
}
