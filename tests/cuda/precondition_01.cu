#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Check that dealii::SolverCG works with CUDAWrappers::SparseMatrix

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_sparse_matrix.h>
#include <deal.II/lac/precondition.h>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/lac/solver_cg.h>
#include <deal.II/lac/solver_control.h>
#include <deal.II/lac/vector.h>

#include <memory>

#include "../testmatrix.h"
#include "../tests.h"

DEAL_II_NAMESPACE_OPEN

namespace CUDAWrappers
{
  /** \addtogroup CUDAWrappers
   *  @{
   */

  /**
   * Template wrapper for cusparse<t>csric02
   * (https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csric02).
   * This function performs the solve phase of the computing the
   * incomplete-Cholesky factorization with 0 fill-in and no pivoting.
   */
  template <typename Number>
  hipsparseStatus_t
  cusparseXcsric02(hipsparseHandle_t         handle,
                   int                      m,
                   int                      nnz,
                   const hipsparseMatDescr_t descrA,
                   Number *                 csrValA_valM,
                   const int *              csrRowPtrA,
                   const int *              csrColIndA,
                   csric02Info_t            info,
                   hipsparseSolvePolicy_t    policy,
                   void *                   pBuffer)
  {
    AssertThrow(false, ExcNotImplemented());
    return HIPSPARSE_STATUS_INVALID_VALUE;
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02<float>(hipsparseHandle_t         handle,
                          int                      m,
                          int                      nnz,
                          const hipsparseMatDescr_t descrA,
                          float *                  csrValA_valM,
                          const int *              csrRowPtrA,
                          const int *              csrColIndA,
                          csric02Info_t            info,
                          hipsparseSolvePolicy_t    policy,
                          void *                   pBuffer)
  {
    return hipsparseScsric02(handle,
                            m,
                            nnz,
                            descrA,
                            csrValA_valM,
                            csrRowPtrA,
                            csrColIndA,
                            info,
                            policy,
                            pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02<double>(hipsparseHandle_t         handle,
                           int                      m,
                           int                      nnz,
                           const hipsparseMatDescr_t descrA,
                           double *                 csrValA_valM,
                           const int *              csrRowPtrA,
                           const int *              csrColIndA,
                           csric02Info_t            info,
                           hipsparseSolvePolicy_t    policy,
                           void *                   pBuffer)
  {
    return hipsparseDcsric02(handle,
                            m,
                            nnz,
                            descrA,
                            csrValA_valM,
                            csrRowPtrA,
                            csrColIndA,
                            info,
                            policy,
                            pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02<hipComplex>(hipsparseHandle_t         handle,
                              int                      m,
                              int                      nnz,
                              const hipsparseMatDescr_t descrA,
                              hipComplex *              csrValA_valM,
                              const int *              csrRowPtrA,
                              const int *              csrColIndA,
                              csric02Info_t            info,
                              hipsparseSolvePolicy_t    policy,
                              void *                   pBuffer)
  {
    return hipsparseCcsric02(handle,
                            m,
                            nnz,
                            descrA,
                            csrValA_valM,
                            csrRowPtrA,
                            csrColIndA,
                            info,
                            policy,
                            pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02<hipDoubleComplex>(hipsparseHandle_t         handle,
                                    int                      m,
                                    int                      nnz,
                                    const hipsparseMatDescr_t descrA,
                                    hipDoubleComplex *        csrValA_valM,
                                    const int *              csrRowPtrA,
                                    const int *              csrColIndA,
                                    csric02Info_t            info,
                                    hipsparseSolvePolicy_t    policy,
                                    void *                   pBuffer)
  {
    return hipsparseZcsric02(handle,
                            m,
                            nnz,
                            descrA,
                            csrValA_valM,
                            csrRowPtrA,
                            csrColIndA,
                            info,
                            policy,
                            pBuffer);
  }


  /**
   * Template wrapper for cusparse<t>csrsv2_solve
   *(https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrsv2_solve).
   * This function performs the solve phase of csrsv2, a new sparse triangular
   *linear system op(A)*y = alpha*x.
   */
  template <typename Number>
  hipsparseStatus_t
  cusparseXcsrsv2_solve(hipsparseHandle_t         handle,
                        hipsparseOperation_t      transA,
                        int                      m,
                        int                      nnz,
                        const Number *           alpha,
                        const hipsparseMatDescr_t descra,
                        const Number *           csrValA,
                        const int *              csrRowPtrA,
                        const int *              csrColIndA,
                        csrsv2Info_t             info,
                        const Number *           x,
                        Number *                 y,
                        hipsparseSolvePolicy_t    policy,
                        void *                   pBuffer)
  {
    AssertThrow(false, ExcNotImplemented());
    return HIPSPARSE_STATUS_INVALID_VALUE;
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_solve<float>(hipsparseHandle_t         handle,
                               hipsparseOperation_t      transA,
                               int                      m,
                               int                      nnz,
                               const float *            alpha,
                               const hipsparseMatDescr_t descra,
                               const float *            csrValA,
                               const int *              csrRowPtrA,
                               const int *              csrColIndA,
                               csrsv2Info_t             info,
                               const float *            x,
                               float *                  y,
                               hipsparseSolvePolicy_t    policy,
                               void *                   pBuffer)
  {
    return hipsparseScsrsv2_solve(handle,
                                 transA,
                                 m,
                                 nnz,
                                 alpha,
                                 descra,
                                 csrValA,
                                 csrRowPtrA,
                                 csrColIndA,
                                 info,
                                 x,
                                 y,
                                 policy,
                                 pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_solve<double>(hipsparseHandle_t         handle,
                                hipsparseOperation_t      transA,
                                int                      m,
                                int                      nnz,
                                const double *           alpha,
                                const hipsparseMatDescr_t descra,
                                const double *           csrValA,
                                const int *              csrRowPtrA,
                                const int *              csrColIndA,
                                csrsv2Info_t             info,
                                const double *           x,
                                double *                 y,
                                hipsparseSolvePolicy_t    policy,
                                void *                   pBuffer)
  {
    return hipsparseDcsrsv2_solve(handle,
                                 transA,
                                 m,
                                 nnz,
                                 alpha,
                                 descra,
                                 csrValA,
                                 csrRowPtrA,
                                 csrColIndA,
                                 info,
                                 x,
                                 y,
                                 policy,
                                 pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_solve<hipComplex>(hipsparseHandle_t         handle,
                                   hipsparseOperation_t      transA,
                                   int                      m,
                                   int                      nnz,
                                   const hipComplex *        alpha,
                                   const hipsparseMatDescr_t descra,
                                   const hipComplex *        csrValA,
                                   const int *              csrRowPtrA,
                                   const int *              csrColIndA,
                                   csrsv2Info_t             info,
                                   const hipComplex *        x,
                                   hipComplex *              y,
                                   hipsparseSolvePolicy_t    policy,
                                   void *                   pBuffer)
  {
    return hipsparseCcsrsv2_solve(handle,
                                 transA,
                                 m,
                                 nnz,
                                 alpha,
                                 descra,
                                 csrValA,
                                 csrRowPtrA,
                                 csrColIndA,
                                 info,
                                 x,
                                 y,
                                 policy,
                                 pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_solve<hipDoubleComplex>(hipsparseHandle_t         handle,
                                         hipsparseOperation_t      transA,
                                         int                      m,
                                         int                      nnz,
                                         const hipDoubleComplex *  alpha,
                                         const hipsparseMatDescr_t descra,
                                         const hipDoubleComplex *  csrValA,
                                         const int *              csrRowPtrA,
                                         const int *              csrColIndA,
                                         csrsv2Info_t             info,
                                         const hipDoubleComplex *  x,
                                         hipDoubleComplex *        y,
                                         hipsparseSolvePolicy_t    policy,
                                         void *                   pBuffer)
  {
    return hipsparseZcsrsv2_solve(handle,
                                 transA,
                                 m,
                                 nnz,
                                 alpha,
                                 descra,
                                 csrValA,
                                 csrRowPtrA,
                                 csrColIndA,
                                 info,
                                 x,
                                 y,
                                 policy,
                                 pBuffer);
  }


  /**
   * Template wrapper for cusparse<t>csrsv2_analysis
   * (https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrsv2_analysis).
   * This function performs the analysis phase of csrsv2, a new sparse
   * triangular linear system op(A)*y = alpha*x.
   */
  template <typename Number>
  hipsparseStatus_t
  cusparseXcsrsv2_analysis(hipsparseHandle_t         handle,
                           hipsparseOperation_t      transA,
                           int                      m,
                           int                      nnz,
                           const hipsparseMatDescr_t descrA,
                           const Number *           csrValA,
                           const int *              csrRowPtrA,
                           const int *              csrColIndA,
                           csrsv2Info_t             info,
                           hipsparseSolvePolicy_t    policy,
                           void *                   pBuffer)
  {
    AssertThrow(false, ExcNotImplemented());
    return HIPSPARSE_STATUS_INVALID_VALUE;
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_analysis<float>(hipsparseHandle_t         handle,
                                  hipsparseOperation_t      transA,
                                  int                      m,
                                  int                      nnz,
                                  const hipsparseMatDescr_t descrA,
                                  const float *            csrValA,
                                  const int *              csrRowPtrA,
                                  const int *              csrColIndA,
                                  csrsv2Info_t             info,
                                  hipsparseSolvePolicy_t    policy,
                                  void *                   pBuffer)
  {
    return hipsparseScsrsv2_analysis(handle,
                                    transA,
                                    m,
                                    nnz,
                                    descrA,
                                    csrValA,
                                    csrRowPtrA,
                                    csrColIndA,
                                    info,
                                    policy,
                                    pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_analysis<double>(hipsparseHandle_t         handle,
                                   hipsparseOperation_t      transA,
                                   int                      m,
                                   int                      nnz,
                                   const hipsparseMatDescr_t descrA,
                                   const double *           csrValA,
                                   const int *              csrRowPtrA,
                                   const int *              csrColIndA,
                                   csrsv2Info_t             info,
                                   hipsparseSolvePolicy_t    policy,
                                   void *                   pBuffer)
  {
    return hipsparseDcsrsv2_analysis(handle,
                                    transA,
                                    m,
                                    nnz,
                                    descrA,
                                    csrValA,
                                    csrRowPtrA,
                                    csrColIndA,
                                    info,
                                    policy,
                                    pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_analysis<hipComplex>(hipsparseHandle_t         handle,
                                      hipsparseOperation_t      transA,
                                      int                      m,
                                      int                      nnz,
                                      const hipsparseMatDescr_t descrA,
                                      const hipComplex *        csrValA,
                                      const int *              csrRowPtrA,
                                      const int *              csrColIndA,
                                      csrsv2Info_t             info,
                                      hipsparseSolvePolicy_t    policy,
                                      void *                   pBuffer)
  {
    return hipsparseCcsrsv2_analysis(handle,
                                    transA,
                                    m,
                                    nnz,
                                    descrA,
                                    csrValA,
                                    csrRowPtrA,
                                    csrColIndA,
                                    info,
                                    policy,
                                    pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_analysis<hipDoubleComplex>(hipsparseHandle_t         handle,
                                            hipsparseOperation_t      transA,
                                            int                      m,
                                            int                      nnz,
                                            const hipsparseMatDescr_t descrA,
                                            const hipDoubleComplex *  csrValA,
                                            const int *              csrRowPtrA,
                                            const int *              csrColIndA,
                                            csrsv2Info_t             info,
                                            hipsparseSolvePolicy_t    policy,
                                            void *                   pBuffer)
  {
    return hipsparseZcsrsv2_analysis(handle,
                                    transA,
                                    m,
                                    nnz,
                                    descrA,
                                    csrValA,
                                    csrRowPtrA,
                                    csrColIndA,
                                    info,
                                    policy,
                                    pBuffer);
  }



  /**
   * Template wrapper for cusparse<t>csric02_analysis
   * (https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csric02_analysis).
   * This function performs the analysis phase of the incomplete-Cholesky
   * factorization with 0 fill-in and no pivoting.
   */
  template <typename Number>
  hipsparseStatus_t
  cusparseXcsric02_analysis(hipsparseHandle_t         handle,
                            int                      m,
                            int                      nnz,
                            const hipsparseMatDescr_t descrA,
                            const Number *           csrValA,
                            const int *              csrRowPtrA,
                            const int *              csrColIndA,
                            csric02Info_t            info,
                            hipsparseSolvePolicy_t    policy,
                            void *                   pBuffer)
  {
    AssertThrow(false, ExcNotImplemented());
    return HIPSPARSE_STATUS_INVALID_VALUE;
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02_analysis<float>(hipsparseHandle_t         handle,
                                   int                      m,
                                   int                      nnz,
                                   const hipsparseMatDescr_t descrA,
                                   const float *            csrValA,
                                   const int *              csrRowPtrA,
                                   const int *              csrColIndA,
                                   csric02Info_t            info,
                                   hipsparseSolvePolicy_t    policy,
                                   void *                   pBuffer)
  {
    return hipsparseScsric02_analysis(handle,
                                     m,
                                     nnz,
                                     descrA,
                                     csrValA,
                                     csrRowPtrA,
                                     csrColIndA,
                                     info,
                                     policy,
                                     pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02_analysis<double>(hipsparseHandle_t         handle,
                                    int                      m,
                                    int                      nnz,
                                    const hipsparseMatDescr_t descrA,
                                    const double *           csrValA,
                                    const int *              csrRowPtrA,
                                    const int *              csrColIndA,
                                    csric02Info_t            info,
                                    hipsparseSolvePolicy_t    policy,
                                    void *                   pBuffer)
  {
    return hipsparseDcsric02_analysis(handle,
                                     m,
                                     nnz,
                                     descrA,
                                     csrValA,
                                     csrRowPtrA,
                                     csrColIndA,
                                     info,
                                     policy,
                                     pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02_analysis<hipComplex>(hipsparseHandle_t         handle,
                                       int                      m,
                                       int                      nnz,
                                       const hipsparseMatDescr_t descrA,
                                       const hipComplex *        csrValA,
                                       const int *              csrRowPtrA,
                                       const int *              csrColIndA,
                                       csric02Info_t            info,
                                       hipsparseSolvePolicy_t    policy,
                                       void *                   pBuffer)
  {
    return hipsparseCcsric02_analysis(handle,
                                     m,
                                     nnz,
                                     descrA,
                                     csrValA,
                                     csrRowPtrA,
                                     csrColIndA,
                                     info,
                                     policy,
                                     pBuffer);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02_analysis<hipDoubleComplex>(hipsparseHandle_t         handle,
                                             int                      m,
                                             int                      nnz,
                                             const hipsparseMatDescr_t descrA,
                                             const hipDoubleComplex *  csrValA,
                                             const int *           csrRowPtrA,
                                             const int *           csrColIndA,
                                             csric02Info_t         info,
                                             hipsparseSolvePolicy_t policy,
                                             void *                pBuffer)
  {
    return hipsparseZcsric02_analysis(handle,
                                     m,
                                     nnz,
                                     descrA,
                                     csrValA,
                                     csrRowPtrA,
                                     csrColIndA,
                                     info,
                                     policy,
                                     pBuffer);
  }


  /**
   * Template wrapper for cusparse<t>csrsv2_bufferSize
   * (https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrsv2_bufferSize).
   * This function returns the size of the buffer used in csrsv2, a new sparse
   * triangular linear system op(A)*y = alpha*x.
   */
  template <typename Number>
  hipsparseStatus_t
  cusparseXcsrsv2_bufferSize(hipsparseHandle_t         handle,
                             hipsparseOperation_t      transA,
                             int                      m,
                             int                      nnz,
                             const hipsparseMatDescr_t descrA,
                             Number *                 csrValA,
                             const int *              csrRowPtrA,
                             const int *              csrColIndA,
                             csrsv2Info_t             info,
                             int *                    pBufferSizeInBytes)
  {
    AssertThrow(false, ExcNotImplemented());
    return HIPSPARSE_STATUS_INVALID_VALUE;
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_bufferSize<float>(hipsparseHandle_t         handle,
                                    hipsparseOperation_t      transA,
                                    int                      m,
                                    int                      nnz,
                                    const hipsparseMatDescr_t descrA,
                                    float *                  csrValA,
                                    const int *              csrRowPtrA,
                                    const int *              csrColIndA,
                                    csrsv2Info_t             info,
                                    int *                    pBufferSizeInBytes)
  {
    return hipsparseScsrsv2_bufferSize(handle,
                                      transA,
                                      m,
                                      nnz,
                                      descrA,
                                      csrValA,
                                      csrRowPtrA,
                                      csrColIndA,
                                      info,
                                      pBufferSizeInBytes);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_bufferSize<double>(hipsparseHandle_t         handle,
                                     hipsparseOperation_t      transA,
                                     int                      m,
                                     int                      nnz,
                                     const hipsparseMatDescr_t descrA,
                                     double *                 csrValA,
                                     const int *              csrRowPtrA,
                                     const int *              csrColIndA,
                                     csrsv2Info_t             info,
                                     int *pBufferSizeInBytes)
  {
    return hipsparseDcsrsv2_bufferSize(handle,
                                      transA,
                                      m,
                                      nnz,
                                      descrA,
                                      csrValA,
                                      csrRowPtrA,
                                      csrColIndA,
                                      info,
                                      pBufferSizeInBytes);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_bufferSize<hipComplex>(hipsparseHandle_t         handle,
                                        hipsparseOperation_t      transA,
                                        int                      m,
                                        int                      nnz,
                                        const hipsparseMatDescr_t descrA,
                                        hipComplex *              csrValA,
                                        const int *              csrRowPtrA,
                                        const int *              csrColIndA,
                                        csrsv2Info_t             info,
                                        int *pBufferSizeInBytes)
  {
    return hipsparseCcsrsv2_bufferSize(handle,
                                      transA,
                                      m,
                                      nnz,
                                      descrA,
                                      csrValA,
                                      csrRowPtrA,
                                      csrColIndA,
                                      info,
                                      pBufferSizeInBytes);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsrsv2_bufferSize<hipDoubleComplex>(hipsparseHandle_t         handle,
                                              hipsparseOperation_t      transA,
                                              int                      m,
                                              int                      nnz,
                                              const hipsparseMatDescr_t descrA,
                                              hipDoubleComplex *        csrValA,
                                              const int *  csrRowPtrA,
                                              const int *  csrColIndA,
                                              csrsv2Info_t info,
                                              int *        pBufferSizeInBytes)
  {
    return hipsparseZcsrsv2_bufferSize(handle,
                                      transA,
                                      m,
                                      nnz,
                                      descrA,
                                      csrValA,
                                      csrRowPtrA,
                                      csrColIndA,
                                      info,
                                      pBufferSizeInBytes);
  }



  /**
   * Template wrapper for cusparse<t>csric02_bufferSize
   * (https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csric02_bufferSize).
   *This function returns size of buffer used in computing the
   *incomplete-Cholesky factorization with 0 fill-in and no pivoting.
   */
  template <typename Number>
  hipsparseStatus_t
  cusparseXcsric02_bufferSize(hipsparseHandle_t         handle,
                              int                      m,
                              int                      nnz,
                              const hipsparseMatDescr_t descrA,
                              Number *                 csrValA,
                              const int *              csrRowPtrA,
                              const int *              csrColIndA,
                              csric02Info_t            info,
                              int *                    pBufferSizeInBytes)
  {
    AssertThrow(false, ExcNotImplemented());
    return HIPSPARSE_STATUS_INVALID_VALUE;
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02_bufferSize<float>(hipsparseHandle_t         handle,
                                     int                      m,
                                     int                      nnz,
                                     const hipsparseMatDescr_t descrA,
                                     float *                  csrValA,
                                     const int *              csrRowPtrA,
                                     const int *              csrColIndA,
                                     csric02Info_t            info,
                                     int *pBufferSizeInBytes)
  {
    return hipsparseScsric02_bufferSize(handle,
                                       m,
                                       nnz,
                                       descrA,
                                       csrValA,
                                       csrRowPtrA,
                                       csrColIndA,
                                       info,
                                       pBufferSizeInBytes);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02_bufferSize<double>(hipsparseHandle_t         handle,
                                      int                      m,
                                      int                      nnz,
                                      const hipsparseMatDescr_t descrA,
                                      double *                 csrValA,
                                      const int *              csrRowPtrA,
                                      const int *              csrColIndA,
                                      csric02Info_t            info,
                                      int *pBufferSizeInBytes)
  {
    return hipsparseDcsric02_bufferSize(handle,
                                       m,
                                       nnz,
                                       descrA,
                                       csrValA,
                                       csrRowPtrA,
                                       csrColIndA,
                                       info,
                                       pBufferSizeInBytes);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02_bufferSize<hipComplex>(hipsparseHandle_t         handle,
                                         int                      m,
                                         int                      nnz,
                                         const hipsparseMatDescr_t descrA,
                                         hipComplex *              csrValA,
                                         const int *              csrRowPtrA,
                                         const int *              csrColIndA,
                                         csric02Info_t            info,
                                         int *pBufferSizeInBytes)
  {
    return hipsparseCcsric02_bufferSize(handle,
                                       m,
                                       nnz,
                                       descrA,
                                       csrValA,
                                       csrRowPtrA,
                                       csrColIndA,
                                       info,
                                       pBufferSizeInBytes);
  }

  template <>
  hipsparseStatus_t
  cusparseXcsric02_bufferSize<hipDoubleComplex>(hipsparseHandle_t         handle,
                                               int                      m,
                                               int                      nnz,
                                               const hipsparseMatDescr_t descrA,
                                               hipDoubleComplex *        csrValA,
                                               const int *   csrRowPtrA,
                                               const int *   csrColIndA,
                                               csric02Info_t info,
                                               int *         pBufferSizeInBytes)
  {
    return hipsparseZcsric02_bufferSize(handle,
                                       m,
                                       nnz,
                                       descrA,
                                       csrValA,
                                       csrRowPtrA,
                                       csrColIndA,
                                       info,
                                       pBufferSizeInBytes);
  }
  /**
   * @}
   */
} // namespace CUDAWrappers

DEAL_II_NAMESPACE_CLOSE

namespace
{
  template <typename Number>
  void
  delete_device_vector(Number *device_ptr) noexcept
  {
    const hipError_t error_code = hipFree(device_ptr);
    (void)error_code;
    AssertNothrow(error_code == hipSuccess,
                  dealii::ExcCudaError(hipGetErrorString(error_code)));
  }
  template <typename Number>
  Number *
  allocate_device_vector(const std::size_t size)
  {
    Number *device_ptr;
    Utilities::CUDA::malloc(device_ptr, size);
    return device_ptr;
  }
} // namespace

namespace dealii
{
  namespace CUDAWrappers
  {
    template <typename Number>
    class PreconditionIC
    {
    public:
      using size_type = int;

      struct AdditionalData
      {
        AdditionalData(bool use_level_analysis = true);

        bool use_level_analysis;
      };

      PreconditionIC(const Utilities::CUDA::Handle &handle);

      PreconditionIC(const PreconditionIC<Number> &) = delete;

      PreconditionIC &
      operator=(const PreconditionIC<Number> &) = delete;

      ~PreconditionIC();

      void
      initialize(const SparseMatrix<Number> &matrix,
                 const AdditionalData &additional_data = AdditionalData());

      void
      vmult(LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
            const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const;

      void
      Tvmult(LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
             const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const;

      size_type
      m() const;

      size_type
      n() const;

    private:
      hipsparseHandle_t cusparse_handle;

      hipsparseMatDescr_t descr_M;
      hipsparseMatDescr_t descr_L;
      csric02Info_t      info_M;
      csrsv2Info_t       info_L;
      csrsv2Info_t       info_Lt;

      const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
      const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;

      std::unique_ptr<Number[], void (*)(Number *)> P_val_dev;
      const int *                                   P_row_ptr_dev;
      const int *                                   P_column_index_dev;
      std::unique_ptr<Number[], void (*)(Number *)> tmp_dev;

      std::unique_ptr<void, void (*)(void *)> buffer_dev;

      hipsparseSolvePolicy_t policy_L;
      hipsparseSolvePolicy_t policy_Lt;

      int n_rows;
      int n_nonzero_elements;
    };

    template <typename Number>
    PreconditionIC<Number>::AdditionalData::AdditionalData(
      bool use_level_analysis_)
      : use_level_analysis(use_level_analysis_)
    {}



    template <typename Number>
    PreconditionIC<Number>::PreconditionIC(
      const Utilities::CUDA::Handle &handle)
      : cusparse_handle(handle.cusparse_handle)
      , P_val_dev(nullptr, delete_device_vector<Number>)
      , P_row_ptr_dev(nullptr)
      , P_column_index_dev(nullptr)
      , tmp_dev(nullptr, delete_device_vector<Number>)
      , buffer_dev(nullptr, delete_device_vector<void>)
      , policy_L(HIPSPARSE_SOLVE_POLICY_USE_LEVEL)
      , policy_Lt(HIPSPARSE_SOLVE_POLICY_USE_LEVEL)
      , n_rows(0)
      , n_nonzero_elements(0)
    {
      hipsparseStatus_t status;
      // step 1: create a descriptor which contains
      // - matrix M is base-0
      // - matrix L is base-0
      // - matrix L is lower triangular
      // - matrix L has non-unit diagonal
      status = hipsparseCreateMatDescr(&descr_M);
      AssertCusparse(status);
      status = hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
      AssertCusparse(status);
      status = hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);
      AssertCusparse(status);

      status = hipsparseCreateMatDescr(&descr_L);
      AssertCusparse(status);
      status = hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
      AssertCusparse(status);
      status = hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
      AssertCusparse(status);
      status = hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
      AssertCusparse(status);
      status = hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);
      AssertCusparse(status);

      // step 2: create a empty info structure
      // we need one info for csric02 and two info's for csrsv2
      status = hipsparseCreateCsric02Info(&info_M);
      AssertCusparse(status);
      status = hipsparseCreateCsrsv2Info(&info_L);
      AssertCusparse(status);
      status = hipsparseCreateCsrsv2Info(&info_Lt);
      AssertCusparse(status);
    }

    template <typename Number>
    PreconditionIC<Number>::~PreconditionIC()
    {
      // step 8: free resources
      hipsparseStatus_t status = hipsparseDestroyMatDescr(descr_M);
      AssertNothrowCusparse(status);

      status = hipsparseDestroyMatDescr(descr_L);
      AssertNothrowCusparse(status);

      status = hipsparseDestroyCsric02Info(info_M);
      AssertNothrowCusparse(status);

      status = hipsparseDestroyCsrsv2Info(info_L);
      AssertNothrowCusparse(status);

      status = hipsparseDestroyCsrsv2Info(info_Lt);
      AssertNothrowCusparse(status);
    }



    template <typename Number>
    void
    PreconditionIC<Number>::initialize(const SparseMatrix<Number> &A,
                                       const AdditionalData &additional_data)
    {
      if (additional_data.use_level_analysis)
        {
          policy_L  = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
          policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        }
      else
        {
          policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
          policy_Lt = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
        }


      n_rows             = A.m();
      n_nonzero_elements = A.n_nonzero_elements();
      AssertDimension(A.m(), A.n());

      const auto          cusparse_matrix = A.get_cusparse_matrix();
      const Number *const A_val_dev       = std::get<0>(cusparse_matrix);

      // create a copy of the matrix entries
      P_val_dev.reset(allocate_device_vector<Number>(n_nonzero_elements));
      hipError_t cuda_status            = hipMemcpy(P_val_dev.get(),
                                           A_val_dev,
                                           n_nonzero_elements * sizeof(Number),
                                           hipMemcpyDeviceToDevice);
      P_column_index_dev                 = std::get<1>(cusparse_matrix);
      P_row_ptr_dev                      = std::get<2>(cusparse_matrix);
      const hipsparseMatDescr_t mat_descr = std::get<3>(cusparse_matrix);

      // initializa an internal buffer we need later on
      tmp_dev.reset(allocate_device_vector<Number>(n_rows));

      // step 3: query how much memory used in csric02 and csrsv2, and allocate
      // the buffer
      int              BufferSize_M;
      hipsparseStatus_t status = cusparseXcsric02_bufferSize(cusparse_handle,
                                                            n_rows,
                                                            n_nonzero_elements,
                                                            descr_M,
                                                            P_val_dev.get(),
                                                            P_row_ptr_dev,
                                                            P_column_index_dev,
                                                            info_M,
                                                            &BufferSize_M);
      AssertCusparse(status);

      int BufferSize_L;
      status = cusparseXcsrsv2_bufferSize(cusparse_handle,
                                          trans_L,
                                          n_rows,
                                          n_nonzero_elements,
                                          descr_L,
                                          P_val_dev.get(),
                                          P_row_ptr_dev,
                                          P_column_index_dev,
                                          info_L,
                                          &BufferSize_L);
      AssertCusparse(status);

      int BufferSize_Lt;
      status = cusparseXcsrsv2_bufferSize(cusparse_handle,
                                          trans_Lt,
                                          n_rows,
                                          n_nonzero_elements,
                                          descr_L,
                                          P_val_dev.get(),
                                          P_row_ptr_dev,
                                          P_column_index_dev,
                                          info_Lt,
                                          &BufferSize_Lt);
      AssertCusparse(status);

      const int BufferSize =
        std::max(BufferSize_M, std::max(BufferSize_L, BufferSize_Lt));
      // workaround: since allocate_device_vector needs a type, we pass char
      // which is required to have size 1.
      buffer_dev.reset(static_cast<void *>(
        allocate_device_vector<char>(BufferSize / sizeof(char))));

      // step 4: perform analysis of incomplete Cholesky on M
      //         perform analysis of triangular solve on L
      //         perform analysis of triangular solve on L'
      // The lower triangular part of M has the same sparsity pattern as L, so
      // we can do analysis of csric02 and csrsv2 simultaneously.

      const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
      status = cusparseXcsric02_analysis(cusparse_handle,
                                         n_rows,
                                         n_nonzero_elements,
                                         descr_M,
                                         P_val_dev.get(),
                                         P_row_ptr_dev,
                                         P_column_index_dev,
                                         info_M,
                                         policy_M,
                                         buffer_dev.get());
      AssertCusparse(status);

      int structural_zero;
      status =
        hipsparseXcsric02_zeroPivot(cusparse_handle, info_M, &structural_zero);
      AssertCusparse(status);

      status = cusparseXcsrsv2_analysis(cusparse_handle,
                                        trans_Lt,
                                        n_rows,
                                        n_nonzero_elements,
                                        descr_L,
                                        P_val_dev.get(),
                                        P_row_ptr_dev,
                                        P_column_index_dev,
                                        info_Lt,
                                        policy_Lt,
                                        buffer_dev.get());
      AssertCusparse(status);

      status = cusparseXcsrsv2_analysis(cusparse_handle,
                                        trans_L,
                                        n_rows,
                                        n_nonzero_elements,
                                        descr_L,
                                        P_val_dev.get(),
                                        P_row_ptr_dev,
                                        P_column_index_dev,
                                        info_L,
                                        policy_L,
                                        buffer_dev.get());
      AssertCusparse(status);

      // step 5: M = L * L'
      status = cusparseXcsric02(cusparse_handle,
                                n_rows,
                                n_nonzero_elements,
                                descr_M,
                                P_val_dev.get(),
                                P_row_ptr_dev,
                                P_column_index_dev,
                                info_M,
                                policy_M,
                                buffer_dev.get());
      AssertCusparse(status);

      int numerical_zero;
      status =
        hipsparseXcsric02_zeroPivot(cusparse_handle, info_M, &numerical_zero);
      AssertCusparse(status);
    }



    template <typename Number>
    void
    PreconditionIC<Number>::vmult(
      LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
      const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
    {
      Assert(P_val_dev != nullptr, ExcNotInitialized());
      Assert(P_row_ptr_dev != nullptr, ExcNotInitialized());
      Assert(P_column_index_dev != nullptr, ExcNotInitialized());
      AssertDimension(dst.size(), n_rows);
      AssertDimension(src.size(), n_rows);
      Assert(tmp_dev != nullptr, ExcInternalError());

      const Number *const src_dev = src.get_values();
      Number *const       dst_dev = dst.get_values();
      // step 6: solve L*z = alpha*x
      const double     alpha  = 1.;
      hipsparseStatus_t status = cusparseXcsrsv2_solve(cusparse_handle,
                                                      trans_L,
                                                      n_rows,
                                                      n_nonzero_elements,
                                                      &alpha,
                                                      descr_L,
                                                      P_val_dev.get(),
                                                      P_row_ptr_dev,
                                                      P_column_index_dev,
                                                      info_L,
                                                      src_dev,
                                                      tmp_dev.get(),
                                                      policy_L,
                                                      buffer_dev.get());
      AssertCusparse(status);

      // step 7: solve L'*y = alpha*z
      status = cusparseXcsrsv2_solve(cusparse_handle,
                                     trans_Lt,
                                     n_rows,
                                     n_nonzero_elements,
                                     &alpha,
                                     descr_L,
                                     P_val_dev.get(),
                                     P_row_ptr_dev,
                                     P_column_index_dev,
                                     info_Lt,
                                     tmp_dev.get(),
                                     dst_dev,
                                     policy_Lt,
                                     buffer_dev.get());
      AssertCusparse(status);
    }



    template <typename Number>
    void
    PreconditionIC<Number>::Tvmult(
      LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
      const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
    {
      // the constructed preconditioner is symmetric
      vmult(dst, src);
    }

    template <typename Number>
    PreconditionIC<Number>::size_type
    PreconditionIC<Number>::m() const
    {
      return n_rows;
    }


    template <typename Number>
    PreconditionIC<Number>::size_type
    PreconditionIC<Number>::n() const
    {
      return n_rows;
    }



    template <typename Number>
    void
    apply_preconditioner(const SparseMatrix<Number> &A,
                         const hipsparseHandle_t      cusparse_handle,
                         LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
                         const LinearAlgebra::CUDAWrappers::Vector<Number> &src)
    {
      const Number *const    src_dev = src.get_values();
      Number *               dst_dev = dst.get_values();
      const hipsparseHandle_t handle  = cusparse_handle;

      const auto       cusparse_matrix    = A.get_cusparse_matrix();
      Number *         A_val_dev          = std::get<0>(cusparse_matrix);
      const int *const A_row_ptr_dev      = std::get<2>(cusparse_matrix);
      const int *const A_column_index_dev = std::get<1>(cusparse_matrix);
      const hipsparseMatDescr_t mat_descr  = std::get<3>(cusparse_matrix);

      const unsigned int n_rows             = A.m();
      const unsigned int n_nonzero_elements = A.n_nonzero_elements();

      AssertDimension(dst.size(), src.size());
      AssertDimension(A.m(), src.size());
      AssertDimension(A.n(), src.size());

      std::unique_ptr<Number[], void (*)(Number *)> tmp_dev(
        allocate_device_vector<Number>(dst.size()),
        delete_device_vector<Number>);

      // Suppose that A is a m x m sparse matrix represented by CSR format,
      // Assumption:
      // - handle is already created by hipsparseCreate(),
      // - (A_row_ptr_dev, A_column_index_dev, A_val_dev) is CSR of A on device
      // memory,
      // - src_dev is right hand side vector on device memory,
      // - dst_dev is solution vector on device memory.
      // - tmp_dev is intermediate result on device memory.

      hipsparseMatDescr_t          descr_M = mat_descr;
      hipsparseMatDescr_t          descr_L = mat_descr;
      csric02Info_t               info_M  = 0;
      csrsv2Info_t                info_L  = 0;
      csrsv2Info_t                info_Lt = 0;
      int                         BufferSize_M;
      int                         BufferSize_L;
      int                         BufferSize_Lt;
      int                         BufferSize;
      void *                      buffer_dev = 0;
      int                         structural_zero;
      int                         numerical_zero;
      const double                alpha     = 1.;
      const hipsparseSolvePolicy_t policy_M  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
      const hipsparseSolvePolicy_t policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
      const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
      const hipsparseOperation_t   trans_L   = HIPSPARSE_OPERATION_NON_TRANSPOSE;
      const hipsparseOperation_t   trans_Lt  = HIPSPARSE_OPERATION_TRANSPOSE;

      hipsparseStatus_t status;
      // step 1: create a descriptor which contains
      // - matrix M is base-0
      // - matrix L is base-0
      // - matrix L is lower triangular
      // - matrix L has non-unit diagonal
      status = hipsparseCreateMatDescr(&descr_M);
      AssertCusparse(status);
      status = hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
      AssertCusparse(status);
      status = hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);
      AssertCusparse(status);

      status = hipsparseCreateMatDescr(&descr_L);
      AssertCusparse(status);
      status = hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
      AssertCusparse(status);
      status = hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
      AssertCusparse(status);
      status = hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
      AssertCusparse(status);
      status = hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);
      AssertCusparse(status);

      // step 2: create a empty info structure
      // we need one info for csric02 and two info's for csrsv2
      status = hipsparseCreateCsric02Info(&info_M);
      AssertCusparse(status);
      status = hipsparseCreateCsrsv2Info(&info_L);
      AssertCusparse(status);
      status = hipsparseCreateCsrsv2Info(&info_Lt);
      AssertCusparse(status);

      // step 3: query how much memory used in csric02 and csrsv2, and allocate
      // the buffer
      status = cusparseXcsric02_bufferSize(handle,
                                           n_rows,
                                           n_nonzero_elements,
                                           descr_M,
                                           A_val_dev,
                                           A_row_ptr_dev,
                                           A_column_index_dev,
                                           info_M,
                                           &BufferSize_M);
      AssertCusparse(status);
      status = cusparseXcsrsv2_bufferSize(handle,
                                          trans_L,
                                          n_rows,
                                          n_nonzero_elements,
                                          descr_L,
                                          A_val_dev,
                                          A_row_ptr_dev,
                                          A_column_index_dev,
                                          info_L,
                                          &BufferSize_L);
      AssertCusparse(status);
      status = cusparseXcsrsv2_bufferSize(handle,
                                          trans_Lt,
                                          n_rows,
                                          n_nonzero_elements,
                                          descr_L,
                                          A_val_dev,
                                          A_row_ptr_dev,
                                          A_column_index_dev,
                                          info_Lt,
                                          &BufferSize_Lt);
      AssertCusparse(status);

      BufferSize = max(BufferSize_M, max(BufferSize_L, BufferSize_Lt));

      // buffer_dev returned by hipMalloc is automatically aligned to 128
      // bytes.
      hipError_t status_cuda = hipMalloc((void **)&buffer_dev, BufferSize);
      Assert(hipSuccess == status_cuda, ExcInternalError());

      // step 4: perform analysis of incomplete Cholesky on M
      //         perform analysis of triangular solve on L
      //         perform analysis of triangular solve on L'
      // The lower triangular part of M has the same sparsity pattern as L, so
      // we can do analysis of csric02 and csrsv2 simultaneously.

      status = cusparseXcsric02_analysis(handle,
                                         n_rows,
                                         n_nonzero_elements,
                                         descr_M,
                                         A_val_dev,
                                         A_row_ptr_dev,
                                         A_column_index_dev,
                                         info_M,
                                         policy_M,
                                         buffer_dev);
      AssertCusparse(status);
      status = hipsparseXcsric02_zeroPivot(handle, info_M, &structural_zero);
      if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
        }

      status = cusparseXcsrsv2_analysis(handle,
                                        trans_Lt,
                                        n_rows,
                                        n_nonzero_elements,
                                        descr_L,
                                        A_val_dev,
                                        A_row_ptr_dev,
                                        A_column_index_dev,
                                        info_Lt,
                                        policy_Lt,
                                        buffer_dev);
      AssertCusparse(status);

      status = cusparseXcsrsv2_analysis(handle,
                                        trans_L,
                                        n_rows,
                                        n_nonzero_elements,
                                        descr_L,
                                        A_val_dev,
                                        A_row_ptr_dev,
                                        A_column_index_dev,
                                        info_L,
                                        policy_L,
                                        buffer_dev);
      AssertCusparse(status);

      // step 5: M = L * L'
      status = cusparseXcsric02(handle,
                                n_rows,
                                n_nonzero_elements,
                                descr_M,
                                A_val_dev,
                                A_row_ptr_dev,
                                A_column_index_dev,
                                info_M,
                                policy_M,
                                buffer_dev);
      AssertCusparse(status);
      status = hipsparseXcsric02_zeroPivot(handle, info_M, &numerical_zero);
      if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
        }

      // step 6: solve L*z = x
      status = cusparseXcsrsv2_solve(handle,
                                     trans_L,
                                     n_rows,
                                     n_nonzero_elements,
                                     &alpha,
                                     descr_L,
                                     A_val_dev,
                                     A_row_ptr_dev,
                                     A_column_index_dev,
                                     info_L,
                                     src_dev,
                                     tmp_dev.get(),
                                     policy_L,
                                     buffer_dev);
      AssertCusparse(status);

      // step 7: solve L'*y = z
      status = cusparseXcsrsv2_solve(handle,
                                     trans_Lt,
                                     n_rows,
                                     n_nonzero_elements,
                                     &alpha,
                                     descr_L,
                                     A_val_dev,
                                     A_row_ptr_dev,
                                     A_column_index_dev,
                                     info_Lt,
                                     tmp_dev.get(),
                                     dst_dev,
                                     policy_Lt,
                                     buffer_dev);
      AssertCusparse(status);

      // step 8: free resources
      status_cuda = hipFree(buffer_dev);
      AssertCuda(status_cuda);
      status = hipsparseDestroyMatDescr(descr_M);
      AssertCusparse(status);
      status = hipsparseDestroyMatDescr(descr_L);
      AssertCusparse(status);
      status = hipsparseDestroyCsric02Info(info_M);
      AssertCusparse(status);
      status = hipsparseDestroyCsrsv2Info(info_L);
      AssertCusparse(status);
      status = hipsparseDestroyCsrsv2Info(info_Lt);
      AssertCusparse(status);
    }
  } // namespace CUDAWrappers
} // namespace dealii

void
test(Utilities::CUDA::Handle &cuda_handle)
{
  // Build the sparse matrix on the host
  const unsigned int   problem_size = 10;
  unsigned int         size         = (problem_size - 1) * (problem_size - 1);
  FDMatrix             testproblem(problem_size, problem_size);
  SparsityPattern      structure(size, size, 5);
  SparseMatrix<double> A;
  testproblem.five_point_structure(structure);
  structure.compress();
  A.reinit(structure);
  testproblem.five_point(A);
  A.print(std::cout);

  // Solve on the host
  PreconditionIdentity prec_no;
  SolverControl        control(100, 1.e-10);
  SolverCG<>           cg_host(control);
  Vector<double>       sol_host(size);
  Vector<double>       rhs_host(size);
  for (unsigned int i = 0; i < size; ++i)
    rhs_host[i] = static_cast<double>(i);
  cg_host.solve(A, sol_host, rhs_host, prec_no);

  // Solve on the device
  CUDAWrappers::SparseMatrix<double>          A_dev(cuda_handle, A);
  LinearAlgebra::CUDAWrappers::Vector<double> sol_dev(size);
  LinearAlgebra::CUDAWrappers::Vector<double> rhs_dev(size);
  LinearAlgebra::ReadWriteVector<double>      rw_vector(size);
  for (unsigned int i = 0; i < size; ++i)
    rw_vector[i] = static_cast<double>(i);
  rhs_dev.import(rw_vector, VectorOperation::insert);
  SolverCG<LinearAlgebra::CUDAWrappers::Vector<double>> cg_dev(control);

  A_dev.print(std::cout);
  A_dev.print_formatted(std::cout);
  CUDAWrappers::PreconditionIC<double>          prec_double(cuda_handle);
  CUDAWrappers::PreconditionIC<float>           prec_float(cuda_handle);
  CUDAWrappers::PreconditionIC<hipComplex>       prec_complex_float(cuda_handle);
  CUDAWrappers::PreconditionIC<hipDoubleComplex> prec_complex_double(
    cuda_handle);

  // apply_preconditioner(A_dev, cuda_handle.cusparse_handle, sol_dev, rhs_dev);
  // A_dev.print_formatted(std::cout);
  prec_double.initialize(A_dev);
  A_dev.print_formatted(std::cout);
  // prec_double.vmult(sol_dev, rhs_dev);
  // A_dev.print_formatted(std::cout);
  cg_dev.solve(A_dev, sol_dev, rhs_dev, prec_double);

  // Check the result
  rw_vector.import(sol_dev, VectorOperation::insert);
  for (unsigned int i = 0; i < size; ++i)
    std::cout << rw_vector[i] << " " << sol_host[i] << std::endl;
}

int
main()
{
  initlog();
  deallog.depth_console(0);

  Utilities::CUDA::Handle cuda_handle;
  test(cuda_handle);

  deallog << "OK" << std::endl;

  return 0;
}
