#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2017 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// test for the Partitioner with a smaller ghost index set within a larger one
// regarding the import_from_ghosted_array() calls
#include <deal.II/base/index_set.h>
#include <deal.II/base/partitioner.h>
#include <deal.II/base/utilities.h>

#include <fstream>
#include <iostream>
#include <vector>

#include "../tests.h"

template <typename Number>
void
print_cuda_view(const ArrayView<Number, MemorySpace::CUDA> cuda_view)
{
  std::vector<Number> cpu_values(cuda_view.size());
  Utilities::CUDA::copy_to_host(cuda_view.data(), cpu_values);
  for (Number value : cpu_values)
    deallog << value << " ";
  deallog << std::endl;
}


void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);
  Assert(numproc > 2, ExcNotImplemented());

  const unsigned int set = 50;
  AssertIndexRange(numproc, set - 2);
  const unsigned int      local_size  = set - myid;
  types::global_dof_index global_size = 0;
  types::global_dof_index my_start    = 0;
  for (unsigned int i = 0; i < numproc; ++i)
    {
      global_size += set - i;
      if (i < myid)
        my_start += set - i;
    }

  // each processor owns some indices and all are ghosting elements from three
  // processors (the second). some entries are right around the border between
  // two processors
  IndexSet local_owned(global_size);
  local_owned.add_range(my_start, my_start + local_size);
  IndexSet local_relevant_1(global_size), local_relevant_2(global_size);
  local_relevant_1                          = local_owned;
  types::global_dof_index ghost_indices[10] = {1,
                                               2,
                                               13,
                                               set - 2,
                                               set - 1,
                                               set,
                                               set + 1,
                                               2 * set,
                                               2 * set + 1,
                                               2 * set + 3};
  local_relevant_1.add_indices(&ghost_indices[0], ghost_indices + 10);
  if (myid > 0)
    local_relevant_1.add_range(my_start - 10, my_start);
  if (myid < numproc - 1)
    local_relevant_1.add_range(my_start + local_size,
                               my_start + local_size + 10);

  local_relevant_2 = local_owned;
  local_relevant_2.add_indices(&ghost_indices[0], ghost_indices + 10);
  if (myid > 0)
    local_relevant_2.add_index(my_start - 10);
  if (myid < numproc - 1)
    local_relevant_2.add_index(my_start + local_size + 9);

  Utilities::MPI::Partitioner v(local_owned, local_relevant_1, MPI_COMM_WORLD);
  Utilities::MPI::Partitioner w(local_owned, MPI_COMM_WORLD);
  w.set_ghost_indices(local_relevant_2, v.ghost_indices());

  IndexSet local_relevant_3(global_size);
  local_relevant_3.add_index(2);
  if (myid > 0 && my_start > 0)
    local_relevant_3.add_range(my_start - 10, my_start);
  Utilities::MPI::Partitioner x(local_owned, MPI_COMM_WORLD);
  x.set_ghost_indices(local_relevant_3, v.ghost_indices());

  // set up a ghost array with some entries
  std::vector<unsigned int> cpu_ghost_array(v.n_ghost_indices(), 1);
  std::unique_ptr<unsigned int[], void (*)(unsigned int *)> ghost_array(
    Utilities::CUDA::allocate_device_data<unsigned int>(cpu_ghost_array.size()),
    Utilities::CUDA::delete_device_data<unsigned int>);
  ArrayView<unsigned int, MemorySpace::CUDA> ghost_array_view(
    ghost_array.get(), cpu_ghost_array.size());
  Utilities::CUDA::copy_to_dev(cpu_ghost_array, ghost_array.get());

  // set up other arrays
  std::unique_ptr<unsigned int[], void (*)(unsigned int *)> locally_owned_array(
    Utilities::CUDA::allocate_device_data<unsigned int>(local_size),
    Utilities::CUDA::delete_device_data<unsigned int>);
  ArrayView<unsigned int, MemorySpace::CUDA> locally_owned_array_view(
    locally_owned_array.get(), local_size);

  std::unique_ptr<unsigned int[], void (*)(unsigned int *)> temp_array(
    Utilities::CUDA::allocate_device_data<unsigned int>(v.n_import_indices()),
    Utilities::CUDA::delete_device_data<unsigned int>);
  ArrayView<unsigned int, MemorySpace::CUDA> temp_array_view(
    temp_array.get(), v.n_import_indices());

  std::vector<MPI_Request> requests;

  // send the full array
  {
    std::unique_ptr<unsigned int[], void (*)(unsigned int *)> ghosts(
      Utilities::CUDA::allocate_device_data<unsigned int>(
        ghost_array_view.size()),
      Utilities::CUDA::delete_device_data<unsigned int>);
    ArrayView<unsigned int, MemorySpace::CUDA> ghosts_view(
      ghosts.get(), ghost_array_view.size());
    const hipError_t cuda_error =
      hipMemcpy(ghosts.get(),
                 ghost_array_view.data(),
                 ghost_array_view.size() * sizeof(unsigned int),
                 hipMemcpyDeviceToDevice);
    AssertCuda(cuda_error);

    v.import_from_ghosted_array_start<unsigned int, MemorySpace::CUDA>(
      VectorOperation::add, 3, ghosts_view, temp_array_view, requests);
    v.import_from_ghosted_array_finish<unsigned int, MemorySpace::CUDA>(
      VectorOperation::add,
      temp_array_view,
      locally_owned_array_view,
      ghosts_view,
      requests);
    // check that the ghost entries are zeroed out in these calls
    deallog << "v ghost entries (should be zero up to index "
            << v.n_ghost_indices() - 1 << "):" << std::endl;
    print_cuda_view(ghosts_view);
  }
  deallog << "From all ghosts: ";
  print_cuda_view(locally_owned_array_view);

  // send only the array in w
  hipError_t cuda_error =
    hipMemset(locally_owned_array_view.data(),
               0,
               locally_owned_array_view.size() * sizeof(unsigned int));
  AssertCuda(cuda_error);
  Assert(temp_array_view.size() >= w.n_import_indices(), ExcInternalError());
  ArrayView<unsigned int, MemorySpace::CUDA> temp_array_view_w(
    temp_array_view.data(), w.n_import_indices());
  {
    std::unique_ptr<unsigned int[], void (*)(unsigned int *)> ghosts(
      Utilities::CUDA::allocate_device_data<unsigned int>(
        ghost_array_view.size()),
      Utilities::CUDA::delete_device_data<unsigned int>);
    ArrayView<unsigned int, MemorySpace::CUDA> ghosts_view(
      ghosts.get(), ghost_array_view.size());
    const hipError_t cuda_error =
      hipMemcpy(ghosts.get(),
                 ghost_array_view.data(),
                 ghost_array_view.size() * sizeof(unsigned int),
                 hipMemcpyDeviceToDevice);
    AssertCuda(cuda_error);

    w.import_from_ghosted_array_start<unsigned int, MemorySpace::CUDA>(
      VectorOperation::add, 3, ghosts_view, temp_array_view_w, requests);
    w.import_from_ghosted_array_finish<unsigned int, MemorySpace::CUDA>(
      VectorOperation::add,
      temp_array_view_w,
      locally_owned_array_view,
      ghosts_view,
      requests);

    // check that the ghost entries are zeroed out in these calls
    deallog << "w ghost entries (should be zero up to index "
            << w.n_ghost_indices() - 1 << "):" << std::endl;
    print_cuda_view(ghosts_view);
  }
  deallog << "From reduced ghosts 1: ";
  print_cuda_view(locally_owned_array_view);

  // send only the array in x
  cuda_error =
    hipMemset(locally_owned_array_view.data(),
               0,
               locally_owned_array_view.size() * sizeof(unsigned int));
  AssertCuda(cuda_error);
  Assert(temp_array_view.size() >= x.n_import_indices(), ExcInternalError());
  ArrayView<unsigned int, MemorySpace::CUDA> temp_array_view_x(
    temp_array_view.data(), x.n_import_indices());
  {
    std::unique_ptr<unsigned int[], void (*)(unsigned int *)> ghosts(
      Utilities::CUDA::allocate_device_data<unsigned int>(
        ghost_array_view.size()),
      Utilities::CUDA::delete_device_data<unsigned int>);
    ArrayView<unsigned int, MemorySpace::CUDA> ghosts_view(
      ghosts.get(), ghost_array_view.size());
    const hipError_t cuda_error =
      hipMemcpy(ghosts.get(),
                 ghost_array_view.data(),
                 ghost_array_view.size() * sizeof(unsigned int),
                 hipMemcpyDeviceToDevice);
    AssertCuda(cuda_error);

    x.import_from_ghosted_array_start<unsigned int, MemorySpace::CUDA>(
      VectorOperation::add, 3, ghosts_view, temp_array_view_x, requests);
    x.import_from_ghosted_array_finish<unsigned int, MemorySpace::CUDA>(
      VectorOperation::add,
      temp_array_view_x,
      locally_owned_array_view,
      ghosts_view,
      requests);

    // check that the ghost entries are zeroed out in these calls
    deallog << "x ghost entries (should be zero up to index "
            << x.n_ghost_indices() << "):" << std::endl;
    print_cuda_view(ghosts_view);
  }
  deallog << "From reduced ghosts 2: ";
  print_cuda_view(locally_owned_array_view);

  // now send a tight array from x and add into the existing entries
  std::vector<unsigned int> cpu_ghosts(x.n_ghost_indices(), 1);
  std::unique_ptr<unsigned int[], void (*)(unsigned int *)> ghosts(
    Utilities::CUDA::allocate_device_data<unsigned int>(cpu_ghosts.size()),
    Utilities::CUDA::delete_device_data<unsigned int>);
  ArrayView<unsigned int, MemorySpace::CUDA> ghosts_view(ghosts.get(),
                                                         cpu_ghosts.size());
  Utilities::CUDA::copy_to_dev(cpu_ghosts, ghosts.get());

  x.import_from_ghosted_array_start<unsigned int, MemorySpace::CUDA>(
    VectorOperation::add, 3, ghosts_view, temp_array_view_x, requests);
  x.import_from_ghosted_array_finish<unsigned int, MemorySpace::CUDA>(
    VectorOperation::add,
    temp_array_view_x,
    locally_owned_array_view,
    ghosts_view,
    requests);
  deallog << "From tight reduced ghosts 2: ";
  print_cuda_view(locally_owned_array_view);
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi(argc, argv);
  MPILogInitAll                    log;
  init_cuda(true);
  test();
}
