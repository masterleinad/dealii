#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------


// check LA::Vector::compress(VectorOperation::min/max) from ghosts

#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <iostream>
#include <vector>

#include "../tests.h"


__global__ void
set_value(double *values_dev, unsigned int index, double val)
{
  values_dev[index] = val;
}


template <typename Number>
double
print_value(Number *values_dev, unsigned int index)
{
  static std::vector<Number> cpu_value(1);
  Utilities::CUDA::copy_to_host(values_dev + index, cpu_value);
  return cpu_value[0];
}



void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  if (myid == 0)
    deallog << "numproc=" << numproc << std::endl;


  // each processor owns 2 indices and all
  // are ghosting element 1 (the second)
  IndexSet local_owned(numproc * 2);
  local_owned.add_range(myid * 2, myid * 2 + 2);
  IndexSet local_relevant(numproc * 2);
  local_relevant = local_owned;
  local_relevant.add_range(1, 2);

  // create vector
  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v(
    local_owned, local_relevant, MPI_COMM_WORLD);
  const auto &partitioner = v.get_partitioner();

  // the read write vector additionally has ghost elements
  IndexSet                               read_write_owned(numproc * 2);
  LinearAlgebra::ReadWriteVector<double> read_write_vector(local_relevant);

  read_write_vector.local_element(0) = myid;
  read_write_vector(1)               = 2. * myid;

  v.import(read_write_vector, VectorOperation::max);
  v.update_ghost_values();

  deallog << myid << ":"
          << "ghost entry after max: "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;

  if (myid == 0)
    read_write_vector(1) = -1.0;

  v.import(read_write_vector, VectorOperation::min);
  v.update_ghost_values();

  deallog << myid << ":"
          << "ghost entry after min: "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;


  if (myid == 0)
    deallog << "OK" << std::endl;
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  MPILogInitAll log;

  init_cuda(true);

  test();
}
