#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Test operator[] and norm_square of cuda_tensor.

#include <deal.II/base/tensor.h>

#include "../tests.h"

template <int rank, int dim, typename Number>
__global__ void multiplication_kernel(Tensor<rank, dim, Number> *t, Tensor<rank, dim, Number> *t1, Tensor<rank, dim, Number> *t2)
{
if (threadIdx.x == 0)
{
t1 = *t*2.;
t2 = 2.*t;
t *= 2.
}
}

template <int rank, int dim, typename Number>
__global__ void division_kernel(Tensor<rank, dim, Number> *t, Tensor<rank, dim, Number> *t1)
{
if (threadIdx.x == 0)
{
t1 = *t/2.;
t /= 2.
}
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<0, dim, Number> *t)
{ 
if (threadIdx.x == 0)
*t = 1.;
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<1, dim, Number> *t)
{ 
  const unsigned int i = threadIdx.x;
  if (i < dim)
      (*t)[i] = i + 1.;
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<2, dim, Number> *t)
{
  const unsigned int i = threadIdx.y;
  const unsigned int j = threadIdx.x;
  if ((i < dim) && (j < dim))
      (*t)[i][j] = j + i * dim + 1.;
}


template <int rank, int dim, typename Number>
void
test_gpu()
{
  const double tolerance = 1.e-8;

  Tensor<rank, dim, Number> *   t_dev;
  Tensor<rank, dim, Number> *   t1_dev;
  Tensor<rank, dim, Number> *   t2_dev;

  Tensor<rank, dim, Number>   t_host;
  Tensor<rank, dim, Number>   t1_host;
  Tensor<rank, dim, Number>   t2_host;

  Tensor<rank, dim, Number> reference_host; 

  // Allocate objects on the device
  hipError_t cuda_error = hipMalloc(&t_dev, sizeof(Tensor<rank, dim, Number>));
  AssertCuda(cuda_error);
  hipError_t cuda_error = hipMalloc(&t1_dev, sizeof(Tensor<rank, dim, Number>));
  AssertCuda(cuda_error);
  hipError_t cuda_error = hipMalloc(&t2_dev, sizeof(Tensor<rank, dim, NUmber>));
  AssertCuda(cuda_error);

  // Initialize
  dim3 block_dim(dim, dim);
  init_kernel<<<1, block_dim>>>(t_dev);
cuda_error =
    hipMemcpy(&reference_host, t_dev, sizeof(Tensor<rank, dim, Number>), hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

// Test multiplication.
  multiplication_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);
  cuda_error =
    hipMemcpy(&t_host, t_dev, sizeof(Tensor<rank, dim, Number>), hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error =
    hipMemcpy(&t1_host, t1_dev, sizeof(Tensor<rank, dim, Number>), hipMemcpyDeviceToHost);
  cuda_error =
    hipMemcpy(&t2_host, t2_dev, sizeof(Tensor<rank, dim, Number>), hipMemcpyDeviceToHost);

reference_host *=2;
AssertThrow((t_host-reference_host).norm()< tolerance, ExcInternalError());
AssertThrow((t1_host-reference_host).norm()< tolerance, ExcInternalError());
AssertThrow((t2_host-reference_host).norm()< tolerance, ExcInternalError());

deallog << "multiplication OK" << std::endl;

// Test division.
  devision_kernel<<<1, 1>>>(t_dev, t1_dev);
  cuda_error =
    hipMemcpy(&t_host, t_dev, sizeof(Tensor<rank, dim, Number>), hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error =
    hipMemcpy(&t1_host, t1_dev, sizeof(Tensor<rank, dim, Number>), hipMemcpyDeviceToHost);

reference_host /=2.;
AssertThrow((t_host-reference_host).norm()< tolerance, ExcInternalError());
AssertThrow((t1_host-reference_host).norm()< tolerance, ExcInternalError());
AssertThrow((t2_host-reference_host).norm()< tolerance, ExcInternalError());

deallog << "division OK" << std::endl;

  // Free memory
  cuda_error = hipFree(t_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(t1_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(t2_dev);
  AssertCuda(cuda_error);
}

int
main()
{
initlog();

  init_cuda();

  test_gpu<0,3,double>();
  test_gpu<1,3,double>();
  test_gpu<2,3,double>();
  test_gpu<0,3,float>();
  test_gpu<1,3,float>();
  test_gpu<2,3,float>();
}
