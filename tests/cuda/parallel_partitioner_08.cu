#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2017 - 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// test MPI::Partitioner update_ghosts() and compress() in case we have
// empty owned DoFs

#include <deal.II/base/aligned_vector.h>
#include <deal.II/base/index_set.h>
#include <deal.II/base/mpi.h>
#include <deal.II/base/partitioner.h>

#include "../tests.h"

template <typename Number>
void
print_cuda_view(const ArrayView<Number, MemorySpace::CUDA> cuda_view)
{
  std::vector<Number> cpu_values(cuda_view.size());
  Utilities::CUDA::copy_to_host(cuda_view.data(), cpu_values);
  for (Number value : cpu_values)
    deallog << value << " ";
  deallog << std::endl;
}

__global__ void
set_value(double *values_dev, unsigned int index, double val)
{
  values_dev[index] = val;
}


template <typename Number = double>
void
test()
{
  const unsigned int rank = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);

  // setup index sets
  //                            subset:                    is2
  //                            ghost:       8 9  10 11    is3
  //      rank 0 :  00 01 02 03 04 05 06 07 00 00 00 00
  //      rank 1 :  00 00 00 00 00 00 00 00 00 00 00 00
  //                            ghost:      0  1  2  3     is3
  //                            subset:        1  2        is2
  //
  // expected result update ghosts()
  //
  //      rank 0 :  00 01 02 03 04 05 06 07 00 00 00 00
  //      rank 1 :  00 00 00 00 00 00 00 00 00 01 02 00
  //
  // compress(insert) -- does not change anything but zero ghosts
  //
  // set rank1 ghosts to: 00 10 20 00
  // compress(add)
  //
  //      rank 0 :  00 11 22 03 04 05 06 07 00 00 00 00
  //      rank 1 :  00 00 00 00 00 00 00 00 00 10 20 00


  IndexSet is1(16), is2(16), is3(16);

  if (rank == 0)
    {
      is1.add_range(0, 8);
      // note: empty is2
      is3.add_range(8, 12);
    }
  else if (rank == 1)
    {
      is1.add_range(8, 16);
      is2.add_index(1);
      is2.add_index(2);
      is3.add_range(0, 4);
    }

  // create partitioner
  std::shared_ptr<Utilities::MPI::Partitioner> partitioner(
    new Utilities::MPI::Partitioner(is1, MPI_COMM_WORLD));
  partitioner->set_ghost_indices(is3);
  std::shared_ptr<Utilities::MPI::Partitioner> tight_partitioner(
    new Utilities::MPI::Partitioner(is1, MPI_COMM_WORLD));
  tight_partitioner->set_ghost_indices(is2, is3);

  // create vector
  std::vector<Number> cpu_owned(rank == 0 ? 8 : 0);
  for (unsigned int i = 0; i < cpu_owned.size(); ++i)
    cpu_owned[i] = i;
  std::unique_ptr<Number[], void (*)(Number *)> owned(
    Utilities::CUDA::allocate_device_data<Number>(cpu_owned.size()),
    Utilities::CUDA::delete_device_data<Number>);
  ArrayView<Number, MemorySpace::CUDA> owned_view(owned.get(),
                                                  cpu_owned.size());
  Utilities::CUDA::copy_to_dev(cpu_owned, owned.get());

  std::vector<Number>                           cpu_ghost(4, 0);
  std::unique_ptr<Number[], void (*)(Number *)> ghost(
    Utilities::CUDA::allocate_device_data<Number>(cpu_ghost.size()),
    Utilities::CUDA::delete_device_data<Number>);
  ArrayView<Number, MemorySpace::CUDA> ghost_view(ghost.get(),
                                                  cpu_ghost.size());
  Utilities::CUDA::copy_to_dev(cpu_ghost, ghost.get());

  // update ghost values
  // vector of requests
  std::vector<MPI_Request> requests;
  std::vector<MPI_Request> compress_requests;

  // allocate temporal array
  std::unique_ptr<Number[], void (*)(Number *)> tmp_data(
    Utilities::CUDA::allocate_device_data<Number>(
      tight_partitioner->n_import_indices()),
    Utilities::CUDA::delete_device_data<Number>);
  ArrayView<Number, MemorySpace::CUDA> tmp_data_view(
    tmp_data.get(), tight_partitioner->n_import_indices());

  // begin exchange, and ...
  tight_partitioner->export_to_ghosted_array_start<Number, MemorySpace::CUDA>(
    0, owned_view, tmp_data_view, ghost_view, requests);

  // ... finish exchange
  tight_partitioner->export_to_ghosted_array_finish<Number, MemorySpace::CUDA>(
    ghost_view, requests);

  auto print = [&]() {
    deallog << "owned:" << std::endl;
    print_cuda_view(owned_view);
    deallog << "ghost:" << std::endl;
    print_cuda_view(ghost_view);
  };

  deallog << "update ghosts()" << std::endl;
  print();

  std::unique_ptr<Number[], void (*)(Number *)> import_data(
    Utilities::CUDA::allocate_device_data<Number>(
      tight_partitioner->n_import_indices()),
    Utilities::CUDA::delete_device_data<Number>);
  ArrayView<Number, MemorySpace::CUDA> import_data_view(
    tmp_data.get(), tight_partitioner->n_import_indices());

  // now do insert:
  auto compress = [&](VectorOperation::values operation) {
    const unsigned int counter = 0;
    tight_partitioner
      ->import_from_ghosted_array_start<Number, MemorySpace::CUDA>(
        operation, counter, ghost_view, import_data_view, compress_requests);

    tight_partitioner
      ->import_from_ghosted_array_finish<Number, MemorySpace::CUDA>(
        operation, import_data_view, owned_view, ghost_view, compress_requests);
  };

  deallog << "compress(insert)" << std::endl;
  compress(VectorOperation::insert);
  print();

  if (rank == 1)
    {
      set_value<<<1, 1>>>(ghost.get(), 1, 10);
      set_value<<<1, 1>>>(ghost.get(), 2, 20);
    }

  deallog << "compress(add)" << std::endl;
  compress(VectorOperation::add);
  print();
}

int
main(int argc, char **argv)
{
  using namespace dealii;

  Utilities::MPI::MPI_InitFinalize mpi_initialization(argc, argv, 1);

  MPILogInitAll log;

  init_cuda(true);

  test();

  return 0;
}
