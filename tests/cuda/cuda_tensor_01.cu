#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Test operator[] and norm_square of cuda_tensor.

#include <deal.II/base/logstream.h>
#include <deal.II/base/tensor.h>

#include <fstream>
#include <iomanip>

#include "../tests.h"

void
test_cpu()
{
  double             a[3][3] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
  const unsigned int dim     = 3;
  Tensor<2, dim>     t;
  for (unsigned int i = 0; i < dim; ++i)
    for (unsigned int j = 0; j < dim; ++j)
      t[i][j] = a[i][j];

  Tensor<2, dim> t_copy;
  std::memcpy(&t_copy, &t, 9);
  deallog.push("values");
  for (unsigned int i = 0; i < dim; ++i)
    for (unsigned int j = 0; j < dim; ++j)
      deallog << t_copy[i][j] << std::endl;
  deallog.pop();

  Vector<double> unrolled(9);
  t_copy.unroll(unrolled);
  for (unsigned int i = 0; i < unrolled.size(); ++i)
    std::cout << unrolled(i) << std::endl;

  deallog.push("norm_square");
  deallog << t.norm_square() << std::endl;
  deallog.pop();
}

__global__ void init_kernel(Tensor<2, 3> *t, const unsigned int N)
{
  const unsigned int i = threadIdx.y;
  const unsigned int j = threadIdx.x;
  if ((i < N) && (j < N))
    {
      (*t)[i][j] = j + i * N + 1.;
      printf("%f\n", (*t)[i][j]);
    }
}

template <int rank, int dim>
__global__ void
unroll_kernel(Tensor<rank, dim> *t, double *unrolled)
{
  const unsigned int     i    = threadIdx.x;
  constexpr unsigned int size = Utilities::pow(rank, dim);
  if (i < size)
    unrolled[i] = t[t->unrolled_to_component_indices(i)];
}

__global__ void norm_kernel(Tensor<2, 3> *t, double *norm)
{
  if (threadIdx.x == 0)
    *norm = t->norm_square();
}

void
test_gpu()
{
  const unsigned int dim = 3;
  double *           norm_dev;
  double             norm_host;
  Tensor<2, dim> *   t_dev;

  // Allocate objects on the device
  hipError_t cuda_error = hipMalloc(&t_dev, sizeof(Tensor<2, dim>));
  AssertCuda(cuda_error);
  cuda_error = hipMalloc(&norm_dev, sizeof(double));
  AssertCuda(cuda_error);

  // Launch the kernels.
  dim3 block_dim(dim, dim);
  init_kernel<<<1, block_dim>>>(t_dev, dim);
  norm_kernel<<<1, 1>>>(t_dev, norm_dev);

  // Copy the result to the host
  cuda_error =
    hipMemcpy(&norm_host, norm_dev, sizeof(double), hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  double *unrolled_dev;
  cuda_error = hipMalloc(unrolled_dev, dim * dim * sizeof(double));
  AssertCuda(cuda_error);
  unroll_kernel<<<1, dim * dim>>>(&t_dev, unrolled_dev);
  std::vector<double> unrolled(dim * dim);
  cuda_error = hipMemcpy(&t, t_dev, sizeof(double), hipMemcpyDeviceToHost);
  for (unsigned int i = 0; i < dim; ++i)
    for (unsigned int j = 0; j < dim; ++j)
      {
        deallog << t[i][j] << std::endl;
      }


  // Free memory
  cuda_error = hipFree(t_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(norm_dev);
  AssertCuda(cuda_error);

  // Output result
  deallog.push("norm_square GPU");
  deallog << norm_host << std::endl;
}

int
main()
{
  std::ofstream logfile("output");
  deallog << std::setprecision(5);
  deallog.attach(logfile);

  init_cuda();

  test_cpu();

  test_gpu();
}
