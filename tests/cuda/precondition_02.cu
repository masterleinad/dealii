// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Check that dealii::SolverCG works with CUDAWrappers::SparseMatrix
// and PreconditionILU

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_precondition.h>
#include <deal.II/lac/cuda_sparse_matrix.h>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/lac/solver_cg.h>
#include <deal.II/lac/solver_control.h>

#include "../testmatrix.h"
#include "../tests.h"

template <typename Number>
void
test(Utilities::CUDA::Handle &cuda_handle)
{
  // Build the sparse matrix on the host
  const unsigned int   problem_size = 10;
  unsigned int         size         = (problem_size - 1) * (problem_size - 1);
  FDMatrix             testproblem(problem_size, problem_size);
  SparsityPattern      structure(size, size, 5);
  SparseMatrix<Number> A;
  testproblem.five_point_structure(structure);
  structure.compress();
  A.reinit(structure);
  testproblem.five_point(A);
  A.print(std::cout);

  // Solve on the device
  CUDAWrappers::SparseMatrix<Number>          A_dev(cuda_handle, A);
  LinearAlgebra::CUDAWrappers::Vector<Number> sol_dev(size);
  LinearAlgebra::CUDAWrappers::Vector<Number> rhs_dev(size);
  LinearAlgebra::ReadWriteVector<Number>      rw_vector(size);
  for (unsigned int i = 0; i < size; ++i)
    rw_vector[i] = static_cast<Number>(i);
  rhs_dev.import(rw_vector, VectorOperation::insert);
  SolverControl                                         control(100, 1.e-10);
  SolverCG<LinearAlgebra::CUDAWrappers::Vector<Number>> cg_dev(control);

  CUDAWrappers::PreconditionILU<Number> prec_ilu(cuda_handle);
  prec_ilu.initialize(A_dev);

  cg_dev.solve(A_dev, sol_dev, rhs_dev, prec_ilu);

  // Check the result
  rw_vector.import(sol_dev, VectorOperation::insert);
  for (unsigned int i = 0; i < size; ++i)
    deallog << rw_vector[i] << std::endl;
}

int
main()
{
  initlog();
  deallog.depth_console(0);

  Utilities::CUDA::Handle cuda_handle;
  test<float>(cuda_handle);
  test<double>(cuda_handle);

  deallog << "OK" << std::endl;

  return 0;
}
